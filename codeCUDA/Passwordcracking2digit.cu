#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
/****************************************************************************
  This program gives an example of a poor way to implement a password cracker
  in CUDA C. It is poor because it acheives this with just one thread, which
  is obviously not good given the scale of parallelism available to CUDA
  programs.
 
  The intentions of this program are:
    1) Demonstrate the use of __device__ and __global__ functions
    2) Enable a simulation of password cracking in the absence of library
       with equivalent functionality to libcrypt. The password to be found
       is hardcoded into a function called is_a_match.   

  Compile and run with:
  nvcc -o Passwordcracking2digit Passwordcracking2digit.cu


     To Run:
     ./Passwordcracking2digit > resultscuda_2alp2dig.txt

  Dr Kevan Buckley, University of Wolverhampton, 2018
*****************************************************************************/

/****************************************************************************
  This function returns 1 if the attempt at cracking the password is
  identical to the plain text password string stored in the program.
  Otherwise,it returns 0.
*****************************************************************************/
__device__ int is_a_match(char *attempt) {
  char p1[] = "CV75";
  char p2[] = "FR51";
  char p3[] = "TB51";
  char p4[] = "IS95";

  char *w = attempt;
  char *x = attempt;
  char *y = attempt;
  char *z = attempt;
  char *password1 = p1;
  char *password2 = p2;
  char *password3 = p3;
  char *password4 = p4;

  while(*w == *password1) {
   if(*w == '\0')
    {
    printf("Found password: %s\n",p1);
      break;
    }

    w++;
    password1++;
  }
    
  while(*x == *password2) {
   if(*x == '\0')
    {
    printf("Found password: %s\n",p2);
      break;
}

    x++;
    password2++;
  }

  while(*y == *password3) {
   if(*y == '\0')
    {
    printf("Found password: %s\n",p3);
      break;
    }

    y++;
    password3++;
  }

  while(*z == *password4) {
   if(*z == '\0')
    {
    printf("Found password: %s\n",p4);
      return 1;
    }

    z++;
    password4++;
  }
  return 0;

}
/****************************************************************************
  The kernel function assume that there will be only one thread and uses
  nested loops to generate all possible passwords and test whether they match
  the hidden password.
*****************************************************************************/

__global__ void  kernel() {
char a,b;
 
  char password[5];
  password[4] = '\0';

int i = blockIdx.x+65;
int j = threadIdx.x+65;
char firstValue = i;
char secondValue = j;
    
password[0] = firstValue;
password[1] = secondValue;
    for(a='0'; a<='9'; a++){
      for(b='0'; b<='9'; b++){
            password[2] = a;
            password[3] = b;
          if(is_a_match(password)) {
        //printf("Success");
          }
             else {
         //printf("tried: %s\n", password);          
            }
          }
        } 
      
}
int time_difference(struct timespec *start,
                    struct timespec *finish,
                    long long int *difference) {
  long long int ds =  finish->tv_sec - start->tv_sec;
  long long int dn =  finish->tv_nsec - start->tv_nsec;

  if(dn < 0 ) {
    ds--;
    dn += 1000000000;
  }
  *difference = ds * 1000000000 + dn;
  return !(*difference > 0);
}


int main() {

  struct  timespec start, finish;
  long long int time_elapsed;
  clock_gettime(CLOCK_MONOTONIC, &start);

kernel <<<26,26>>>();
  hipDeviceSynchronize();

  clock_gettime(CLOCK_MONOTONIC, &finish);
  time_difference(&start, &finish, &time_elapsed);
  printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed, (time_elapsed/1.0e9));
  return 0;
}




